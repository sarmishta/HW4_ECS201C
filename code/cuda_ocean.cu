#include <stdio.h>

#define VERSION1
//#define VERSION15
//#define VERSION2
//#define VERSION3

#include "cuda_ocean_kernels.cu"

void Check_CUDA_Error(const char *message)
{
   hipError_t error = hipGetLastError();
   if(error!=hipSuccess) {
      fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
      exit(-1);
   }
}

extern "C++"
void ocean (int **grid, int xdim, int ydim, int timesteps)
{
    /********************* the red-black algortihm (start)************************/
    /*
    In odd timesteps, calculate indeces with - and in even timesteps, calculate indeces with *
    See the example of 6x6 matrix, A represents the corner elements.
        A A A A A A
        A - * - * A
        A * - * - A
        A - * - * A
        A * - * - A
        A A A A A A
    */

    // Set up the GPU datastructure used in the first 3 kernel verisons

    int *d_grid;

    hipMalloc(&d_grid, sizeof(int)*xdim*ydim);
    Check_CUDA_Error("malloc grid failed");

    hipMemcpy(d_grid, &grid[0][0], xdim*ydim*sizeof(int), hipMemcpyHostToDevice);
    Check_CUDA_Error("memcpy grid to device failed");

    #if defined(VERSION3)

    // set up the GPU datastructure for the other kernel versions
    int *red_grid, *black_grid;
    hipMalloc(&red_grid, sizeof(int)*xdim*ydim / 2);
    Check_CUDA_Error("malloc red_grid failed");
    hipMalloc(&black_grid, sizeof(int)*xdim*ydim / 2);
    Check_CUDA_Error("malloc black_grid failed");

    split_array_kernel<<<16,512>>>(d_grid, red_grid, black_grid, xdim, ydim);
    Check_CUDA_Error("split_array_kernel launch failed");

    #endif

    dim3 gridDim(16,1,1);
    dim3 blockDim(128,1,1);

    for (int ts=0; ts<timesteps; ts++) {
        #if defined(VERSION3)
        ocean_kernel<<<gridDim, blockDim>>>(red_grid, black_grid, xdim, ydim, ts%2);
        #else
    	ocean_kernel<<<gridDim, blockDim>>>(d_grid, xdim, ydim, ts%2);
        #endif
    	Check_CUDA_Error("ocean_kernel launch failed");
    }

    #if defined(VERSION3)
    unsplit_array_kernel<<<16,512>>>(d_grid, red_grid, black_grid, xdim, ydim);
    Check_CUDA_Error("unsplit_array_kernel launch failed");
    #endif

    hipMemcpy(&grid[0][0], d_grid, xdim*ydim*sizeof(int), hipMemcpyDeviceToHost);
    Check_CUDA_Error("memcpy grid back failed");

    hipFree(d_grid);


    /////////////////////// the red-black algortihm (end) ///////////////////////////
}
