
#include <hip/hip_runtime.h>



#ifdef VERSION1
__global__ void ocean_kernel(int *grid, int xdim, int ydim, int offset)
{
    int threads = gridDim.x*blockDim.x;
    int threadId  = blockDim.x*blockIdx.x + threadIdx.x;

    if (threads > (xdim-2)*(ydim-2)) {
        threads = (xdim-2)*(ydim-2);
        if (threadId >= threads) {
            return;
        }
    }

    int chunk = (xdim-2)*(ydim-2)/threads;
    int start = threadId * chunk;
    int end = (threadId + 1) * chunk;

    int threadsPerRow = (xdim - 2);

    for (int i=start; i<end; i++) {
        if (offset) {
            if (i%2) continue;
        } else {
            if (!(i%2)) continue;
        }

        int row = i / threadsPerRow;
        int col = i % threadsPerRow;

        int loc = xdim + row * xdim + col;
        if (offset) {
            loc += (row%2) ? 1 : 0;
            loc += 1;
        } else {
            loc += (row%2) ? 0 : 1;
        }
        // printf("Row: %d, Col: %d\n", row, col);
        // printf("loc: %d\n", loc);

        grid[loc] = (grid[loc]
                  + grid[loc - xdim]
                  + grid[loc + xdim]
                  + grid[loc + 1]
                  + grid[loc - 1])
                  / 5;
    }
}
#endif


#ifdef VERSION2
__global__ void ocean_kernel(int *grid, int xdim, int ydim, int offset)
{

    // Your code for task 2
    
        
}
#endif

#ifdef VERSION3

__global__ void split_array_kernel(int *grid, int *red_grid, int *black_grid, int xdim, int ydim)
{
    // This kernel should take the contents of grid and copy all of the red
    // elements into red_grid and all of the black elements into black_grid
}

__global__ void unsplit_array_kernel(int *grid, int *red_grid, int *black_grid, int xdim, int ydim)
{
    // This kernel should take the red_grid and black_grid and copy it back into grid
}

__global__ void ocean_kernel(int *red_grid, int *black_grid, int xdim, int ydim, int offset)
{
    // Your code for task 3
}
#endif
